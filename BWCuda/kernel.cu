#include <stdio.h>
#include <inttypes.h>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>

typedef uint8_t byte;
typedef struct pixel { byte subpixels[3]; } pixel;

__global__ void PixelAverage(pixel* pixels)
{
	byte gray = (pixels[threadIdx.x].subpixels[0] + pixels[threadIdx.x].subpixels[1] + pixels[threadIdx.x].subpixels[2]) / 3;
	pixels[threadIdx.x].subpixels[0] = gray;
	pixels[threadIdx.x].subpixels[1] = gray;
	pixels[threadIdx.x].subpixels[2] = gray;
}

__global__ void PixelLuminosity(pixel* pixels)
{
	byte gray = (0.21 * pixels[threadIdx.x].subpixels[0] + 0.72 * pixels[threadIdx.x].subpixels[1] + 0.07 * pixels[threadIdx.x].subpixels[2]);
	pixels[threadIdx.x].subpixels[0] = gray;
	pixels[threadIdx.x].subpixels[1] = gray;
	pixels[threadIdx.x].subpixels[2] = gray;
}

__global__ void PixelLightness(pixel* pixels)
{
	byte max = (pixels[threadIdx.x].subpixels[0] < pixels[threadIdx.x].subpixels[1]) ? pixels[threadIdx.x].subpixels[1] : pixels[threadIdx.x].subpixels[0];
	max = ((max < pixels[threadIdx.x].subpixels[2]) ? pixels[threadIdx.x].subpixels[2] : max);

	byte min = (pixels[threadIdx.x].subpixels[0] > pixels[threadIdx.x].subpixels[1]) ? pixels[threadIdx.x].subpixels[1] : pixels[threadIdx.x].subpixels[0];
	min = ((min > pixels[threadIdx.x].subpixels[2]) ? pixels[threadIdx.x].subpixels[2] : min);

	byte gray = 0.5 * (max + min);
	pixels[threadIdx.x].subpixels[0] = gray;
	pixels[threadIdx.x].subpixels[1] = gray;
	pixels[threadIdx.x].subpixels[2] = gray;
}

int main()
{
	pixel* pixels;
	int n = 2;
	int size = n * sizeof(pixel);
	hipMallocManaged(&pixels, size);

	pixels[0].subpixels[0] = 42;
	pixels[0].subpixels[1] = 37;
	pixels[0].subpixels[2] = 12;

	pixels[1].subpixels[0] = 236;
	pixels[1].subpixels[1] = 155;
	pixels[1].subpixels[2] = 23;


	PixelLuminosity <<<1, n >>> (pixels);

	hipDeviceSynchronize();

	std::cout << +pixels[0].subpixels[0] << " " << +pixels[0].subpixels[1] << " " << +pixels[0].subpixels[2] << std::endl;
	std::cout << +pixels[1].subpixels[0] << " " << +pixels[1].subpixels[1] << " " << +pixels[1].subpixels[2] << std::endl;

	hipFree(pixels);

	return 0;
}